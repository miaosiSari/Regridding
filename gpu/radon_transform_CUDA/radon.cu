#include "hip/hip_runtime.h"
#include <stdio.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""
#include "time.h"
#include <sys/time.h>
#define SIZE 500   
__global__ static void radon_cuda_core(float *gpuOutput,float *gpuInput,float *gpuAngles,int M,int N,int xOrgin,int yOrgin,int numAngles,int rFirst,int rSize)
{
	const int tid=threadIdx.x;	
	float angle=gpuAngles[tid];
	float *pOutput=gpuOutput+tid*rSize;		
	float sine=sin(angle);
	float cosine=cos(angle);
	int m,n;
 
	for(m=0;m<rSize;m++)
		pOutput[m]=0.0;
 
	float *pInput=gpuInput;
	for(n=0;n<N;n++)
	{
		for(m=0;m<M;m++)
		{
			float value=*pInput++;
			if(value!=0.0)
			{
				value*=0.25;
				float x=n-xOrgin;
				float y=yOrgin-m;
 
				int r1;
				float delta;
					
				float r=(x-0.25)*cosine+(y-0.25)*sine-rFirst;
				r1=(int)r;
				delta=r-r1;
				pOutput[r1]+=value*(1.0-delta);
				pOutput[r1+1]+=value*delta;
 
				r=(x-0.25)*cosine+(y+0.25)*sine-rFirst;
				r1=(int)r;
				delta=r-r1;
				pOutput[r1]+=value*(1.0-delta);
				pOutput[r1+1]+=value*delta;
 
				r=(x+0.25)*cosine+(y+0.25)*sine-rFirst;
				r1=(int)r;
				delta=r-r1;
				pOutput[r1]+=value*(1.0-delta);
				pOutput[r1+1]+=value*delta;
 
				r=(x+0.25)*cosine+(y-0.25)*sine-rFirst;
				r1=(int)r;
				delta=r-r1;
				pOutput[r1]+=value*(1.0-delta);
				pOutput[r1+1]+=value*delta;
			}
		}
	}	
}
static void radon_cuda(float *pPtr, float *iPtr, float *thetaPtr, int M, int N,    
	int xOrigin, int yOrigin, int numAngles, int rFirst, int rSize)
{
	float *gpuInput;
	float *gpuOutput;
	float *gpuAngles;
	hipMalloc((void **)&gpuInput,sizeof(float)*M*N);
	hipMalloc((void **)&gpuOutput,sizeof(float)*numAngles*rSize);
	hipMalloc((void **)&gpuAngles,sizeof(float)*numAngles);
	hipMemcpy(gpuInput,iPtr,sizeof(float)*M*N,hipMemcpyHostToDevice);
	hipMemset(gpuOutput,0,numAngles*rSize);
	hipMemcpy(gpuAngles,thetaPtr,sizeof(float)*numAngles,hipMemcpyHostToDevice);
 
	radon_cuda_core<<<1,numAngles,0>>>(gpuOutput,gpuInput,gpuAngles,M,N,xOrigin,yOrigin,numAngles,rFirst,rSize);
 
	hipMemcpy(pPtr,gpuOutput,sizeof(float)*numAngles*rSize,hipMemcpyDeviceToHost);
 
	hipFree(gpuInput);
	hipFree(gpuOutput);
	hipFree(gpuAngles);
}

long long gettime(){
	struct timeval s1;
	struct timezone s2;
	gettimeofday(&s1, &s2);
	long long time_microsecond = s1.tv_sec*1000000+s1.tv_usec;
	return time_microsecond;
}
 
int main()
{
	int M=SIZE;
	int N=SIZE;
	int xOrigin=((N-1)/2>0)?((N-1)/2):0;
	int yOrigin=((M-1)/2>0)?((M-1)/2):0;
	int temp1=M-1-yOrigin;
	int temp2=N-1-xOrigin;
	int rLast=(int) ceil(sqrt((float) (temp1*temp1+temp2*temp2))) + 1;
	int rFirst=-rLast;
	int rSize=rLast-rFirst+1;
	int numAngles = 181;
	float *thetaPtr= (float*)calloc(numAngles,sizeof(float));
	float *ptr = thetaPtr;
	float deg2rad = 3.141592 / 180.0;
	int k = 0;
	int times = 15;
	for (k = 0; k<numAngles; k++)
		*(ptr++) = numAngles * deg2rad;
	float* I=(float *)calloc(M*N,sizeof(float));
 
	int p = 0;
	for (p =0;p < M; p++)
	{
		for (k = 0;k < N;k++)
		{
			if (k > M/4 && k < M*3/4)
			{
				if (p > M/4 && p < M*3/4)
					I[p * M + k]=1;
				else
					I[p * M + k]=0;
			}
			else
				I[p * M + k]=0;
		}
	}
	float *gpu_result;
	gpu_result=(float *)calloc(numAngles*rSize,sizeof(float));
	memset(gpu_result,0,numAngles*rSize);
	long long start=0, end=0;
	double sum=0.0, DX=0.0;
    double trial[times+1]={0};
	for (int cnt = 0; cnt < times; ++cnt){
		start = gettime();
		radon_cuda(gpu_result, I, thetaPtr, M, N, xOrigin, yOrigin, numAngles, rFirst, rSize);
		end = gettime();
		printf("start=%lld, end=%lld, time=%lld\n", start, end, end-start);
		trial[cnt] = (double)(end-start);
		sum += trial[cnt];
	}
	double average = sum/times;
	printf("Average time=%lf\n", average);
	for (int cnt=0; cnt < times; ++cnt){
        DX += (trial[cnt]-average)*(trial[cnt]-average);
        printf("%lld ",(long long)trial[cnt]);
	}		
        printf("\n");
	printf("DX = %lf\n", DX/times);
	free(I);
	free(thetaPtr);
	free(gpu_result);
	return 0;
}
